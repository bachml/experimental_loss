#include <algorithm>
#include <vector>

#include "caffe/layers/normalize_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {


template <typename Dtype>
void NormalizeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* center_data = temp_center.mutable_gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  caffe_copy(bottom[0]->count(), bottom_data, top_data);
  //step1 : 计算blob  Batch_num x dim 的向量
  for(int i = 0; i < M_; i++) {
    Dtype dot;
    caffe_gpu_dot(K_, bottom_data + K_*i, bottom_data + K_*i, &dot);
    //temp_center[i] = dot;
    caffe_gpu_set(K_, dot, center_data + K_*i);
    caffe_gpu_powx(K_, center_data + K_*i, (Dtype)0.5, center_data + K_*i);
  }
  //step2: 计算forward 
  caffe_gpu_div(M_ * K_, top_data, center_data, top_data);
  
}


template <typename Dtype>
void NormalizeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* center_data = temp_center.gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  
  // for f_d. f_d' = (1 - f_d^2) / ||f||
  //caffe_set(dim*batch_num, (Dtype)1, bottom_diff)
  caffe_gpu_powx(M_*K_, top_data, (Dtype)2, bottom_diff);
  caffe_gpu_add_scalar(M_*K_, (Dtype)-1, bottom_diff);
  caffe_gpu_scal(M_*K_, (Dtype)-1, bottom_diff);
  caffe_gpu_div(M_*K_, bottom_diff, center_data, bottom_diff);
  caffe_gpu_mul(M_*K_, bottom_diff, top_diff, bottom_diff);
}


INSTANTIATE_LAYER_GPU_FUNCS(NormalizeLayer);

}  // namespace caffe
